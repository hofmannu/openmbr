#include "cudaErrorCheck.cuh"

void cudaErrorCheck::checkCudaReturn(const hipError_t err, const std::string errMsg)
{
	if (err != hipSuccess){
		printf("CUDA error string: ");
		printf(hipGetErrorString(err));
		printf("\n");
		printf(errMsg.c_str());
		printf("\n");
		throw "hipError_t";
	}
	return;
}	